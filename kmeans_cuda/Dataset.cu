#include "Dataset.h"
#include <cassert>
#include <iostream>
#include <hip/hip_runtime.h>

// Simple GPU Error Check Macro
#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line) {
    if (code != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(code)
            << " " << file << ":" << line << std::endl;
        exit(code);
    }
}

Dataset::Dataset(const std::vector<float>& inputData, int numPoints, int dimension)
    : h_data_(inputData),
    d_data_(nullptr),
    numPoints_(numPoints),
    dim_(dimension)
{
    // Basic sanity check
    assert(static_cast<int>(h_data_.size()) == numPoints_ * dim_);
    allocateDeviceMemory();
}

Dataset::~Dataset() {
    freeDeviceMemory();
}

int Dataset::getNumPoints() const {
    return numPoints_;
}

int Dataset::getDim() const {
    return dim_;
}

float* Dataset::getDeviceData() const {
    return d_data_;
}

void Dataset::allocateDeviceMemory() {
    size_t dataSize = sizeof(float) * h_data_.size();
    CUDA_CHECK(hipMalloc((void**)&d_data_, dataSize));
    CUDA_CHECK(hipMemcpy(d_data_, h_data_.data(), dataSize, hipMemcpyHostToDevice));
}

void Dataset::freeDeviceMemory() {
    if (d_data_) {
        CUDA_CHECK(hipFree(d_data_));
        d_data_ = nullptr;
    }
}
